#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include "PrecomputedRandAdvance.h"
#include <fstream>
#include <math.h>
#include <chrono>
#include <iomanip>
#include <tuple>
#include "args_parser.h"
#include <limits>
#include <exception>

typedef std::chrono::high_resolution_clock Clock;

#define THREADSPERBLOCK_X 16
#define THREADSPERBLOCK_Y 16
#define THREADSPERBLOCK_2 256

// Return the root of a tree
__device__ unsigned Find(const int* s_buf, unsigned n) {

	unsigned label = s_buf[n];

	while (label - 1 != n) {
		n = label - 1;
		label = s_buf[n];
	}

	return n;

}

// Links together trees containing a and b
__device__ void Union(int* s_buf, unsigned a, unsigned b) {

	bool done;

	do {

		a = Find(s_buf, a);
		b = Find(s_buf, b);

		if (a < b) {
			int old = atomicMin(s_buf + b, a + 1);
			done = (old == b + 1);
			b = old - 1;
		}
		else if (b < a) {
			int old = atomicMin(s_buf + a, b + 1);
			done = (old == a + 1);
			a = old - 1;
		}
		else {
			done = true;
		}

	} while (!done);

}


// Init phase.
// Labels start at value 1, to differentiate them from background, that has value 0.
__global__ void Init(const int* img, int* labels) {

	unsigned row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned col = blockIdx.x * blockDim.x + threadIdx.x;
	int globalSize = gridDim.x * blockDim.x;

	unsigned img_index = row * globalSize + col;
	unsigned labels_index = row * globalSize + col;



	if (img[img_index]) {

		if (row > 0 && img[img_index - globalSize]) {
			labels[labels_index] = labels_index - globalSize + 1;
		}

		else if (col > 0 && img[img_index - 1]) {
			labels[labels_index] = labels_index;
		}

		else {
			labels[labels_index] = labels_index + 1;
		}
	}

}


// Analysis phase.
__global__ void Analyze(int* labels) {

	unsigned row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned col = blockIdx.x * blockDim.x + threadIdx.x;
	int globalSize = gridDim.x * blockDim.x;

	unsigned labels_index = row * globalSize + col;

	unsigned label = labels[labels_index];

	if (label) {

		unsigned index = labels_index;

		while (label - 1 != index) {
			index = label - 1;
			label = labels[index];
		}

		labels[labels_index] = label;
	}
}

__global__ void Reduce(const int* img, int* labels) {

	unsigned row = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned col = blockIdx.x * blockDim.x + threadIdx.x;
	int globalSize = gridDim.x * blockDim.x;

	unsigned img_index = row * globalSize + col;
	unsigned labels_index = row * globalSize + col;


	if (img[img_index]) {

		if (col > 0 && img[img_index - 1]) {
			Union(labels, labels_index, labels_index - 1);
		}
	}
}



__device__ int64_t rawSeedFromChunk(int x, int z)
{
	return (((int64_t)x * (int64_t)341873128712 + (int64_t)z * (int64_t)132897987541) ^ (int64_t)0x5DEECE66D) & ((((int64_t)1 << 48) - 1));
}


__device__ int rand5(int64_t raw_seed, int64_t a, int64_t b)
{
	return (int)((((raw_seed * a + b) & (((int64_t)1 << 48) - 1)) >> 17) % ((int64_t)5));
}


__device__ int precompChunkIndCalcNormal(int x, int y, int z, int nether)
{
	return ((z * 16 + x) * (nether == 1 ? 8 : 4) + ((nether == 1 ? 7 : 3) - y));
}


__device__ int getBedrock(int x, int y, int z, const int64_t* a, const int64_t* b)
{
	if (y == 0) return 1;
	if (y < 0 || y > 4) return 0;
	int precomp_ind = precompChunkIndCalcNormal(x & 15, y - 1, z & 15, 0);
	return (rand5(rawSeedFromChunk(x >> 4, z >> 4), a[precomp_ind], b[precomp_ind]) >= y) ? 1 : 0;
}


__global__ void getBedrockTile(const int64_t* a,  const int64_t* b,  const int* offset, int* outData)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int z = blockIdx.y * blockDim.y + threadIdx.y;
	int globalSize = gridDim.x * blockDim.x;

	outData[globalSize * x + z] = getBedrock(offset[0] + x, 4, offset[1] + z, a, b);
}

__global__ void getFrequency(int* labels, int* freq) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (labels[x] > 0) {
		atomicAdd(&(freq[labels[x] - 1]), 1);
	}
}

__global__ void reduction(int* inData, int* outData, int* outIdData) {
	size_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
	size_t localSize = blockDim.x;
	size_t localId = threadIdx.x;

	__shared__ int localData[THREADSPERBLOCK_2];

	unsigned bits, var = localSize;
	for (bits = 0; var != 0; ++bits) var >>= 1;

	localData[localId] = inData[globalId];
	__syncthreads();

	for (int i = localSize >> 1; i > 0; i >>= 1) {
		if (localId < i) {

			//localData[localId] = max(localData[localId], localData[localId + i]);
			if (localData[localId] > localData[localId + i]) {
				localData[localId + i] = 0; // choose left
			}
			else {
				localData[localId] = localData[localId + i];
				localData[localId + i] = 1; // choose right
			}
		}
		__syncthreads();
	}

	if (localId == 0) {

		int bitsum = 0;
		int nextid;
		for (int i = 0; i < bits - 1; i++) {

			nextid = (1 << i) + bitsum;
			bitsum = (localData[nextid] << i) + bitsum;
		}

		int final_id = localData[nextid] ? nextid : nextid - (localSize >> 1);

		outData[blockIdx.x] = localData[0];
		outIdData[blockIdx.x] = globalId + final_id;
	}


	//8 is 2^3 so we do 3 hops
	//0 0 0 0 0 0 0 0
	//6 | 1 | 0 1 | 0 1 1 1 | -> 1 1 1 -> 8
	//3 | 0 | 1 0 | 0 1 0 0 | -> 0
	//9 | 1 | 0 1 | 1 0 1 0 |
	//7 | 1 | 0 0 | 1 0 1 0 | 0 1 0 0 1 1 1 1 |
	//0 1 2 3 4 5 6 7

	//select 0th bit from column 1 (1) 1 << 0 + 0
	//select 1th bit from column 2 (0) 1 << 1 + 1
	//select 2th bit from column 3 (1) 1 << 2 + 2
	//select 5th bit from column 4 (1) 1 << 3 + 5
	/*int bitsum = 0;
	int nextid;
	for (int i = 0; i < bits - 2; i++) {

		nextid = (1 << i) + bitsum;
		bitsum = (bitsum << 1) + localData[nextid];
	}*/
}

std::ostream& operator<<(std::ostream& os, const std::chrono::microseconds& v) {
	// convert to microseconds
	int64_t us = v.count();

	int h = us / ((int64_t)1000 * (int64_t)1000 * 60 * 60);
	us -= h * ((int64_t)1000 * (int64_t)1000 * 60 * 60);

	int m = us / ((int64_t)1000 * (int64_t)1000 * 60);
	us -= m * ((int64_t)1000 * (int64_t)1000 * 60);

	int s = us / ((int64_t)1000 * (int64_t)1000);
	us -= s * ((int64_t)1000 * (int64_t)1000);

	return os << std::setfill('0') << std::setw(2) << h << ':' << std::setw(2) << m
		<< ':' << std::setw(2) << s;
}

void saveLog(std::string filename, size_t start, size_t end, std::tuple<size_t, int, int> best){
	std::ofstream outfile;
	outfile.open(filename, std::ios_base::app); // append instead of overwrite

	outfile << "Searched: " << start << '-' << end << " Best found: " << std::get<0>(best) << " @ (" << std::get<1>(best) << ", " << std::get<2>(best) << ')' << std::endl;
	outfile.close();
}

std::vector<int> spiral(int n) {
	n++;
	int k = ceil((sqrt(n) - 1) / 2);
	int t = 2 * k + 1;
	int m = t * t;
	t = t - 1;

	if (n >= m - t) {
		return { k - (m - n), -k };
	}
	else { m = m - t; }

	if (n >= m - t) {
		return { -k, -k + (m - n) };
	}
	else { m = m - t; }

	if (n >= m - t) {
		return { -k + (m - n), k };
	}
	else { return { k, k - (m - n - t) }; }
}

int main(int argc, char* argv[])
{
	InputParser args(argc, argv);
	int nDevices;

	hipGetDeviceCount(&nDevices);

	std::cout << nDevices << " devices found" << std::endl;
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		std::cout << i << " : " << prop.name;
		std::cout << " Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
	}
	std::cout << "---------------" << std::endl;

	const bool hasflag_q = args.cmdOptionExists("-q");
	const bool hasflag_b = args.cmdOptionExists("-b");
	const std::string flag_b = args.getCmdOption("-b");

	if ((argc < 3) || (hasflag_b && std::string(argv[2]) != "-b" && std::string(argv[2]) != "-q")){
		throw std::invalid_argument("usage: ./clumpFinderCUDA <start> <end | -b batchsize> [-q]");
	}

	const int len = 8192;
	const size_t start = std::stoi(argv[1]);
	const size_t end = hasflag_b ? UINT_MAX : std::stoi(argv[2]);

	const size_t d = hasflag_b ? std::stoi(flag_b) : 0;
	size_t part_start = hasflag_b ? start : 0;
	size_t part_end = hasflag_b ? start + d - (start % d) : 0;

	std::vector<int> offset = { 0, 0};
	std::vector<int> final((len * len) / 256, 0);
	std::vector<int> finalIds((len * len) / 256, 0);


	hipError_t err;

	int* off_d;
	int* bedrock_d;
	int* freq_d;
	int* final_d;
	int* finalIds_d;
	int* labels_d;

	int64_t* a_d, * b_d;
	err = hipMalloc(&a_d, sizeof(int64_t) * A_OW_112.size());
	err = hipMalloc(&b_d, sizeof(int64_t) * B_OW_112.size());
	err = hipMalloc(&off_d, sizeof(int) * offset.size());
	err = hipMalloc(&bedrock_d, sizeof(int) * len * len);
	err = hipMalloc(&labels_d, sizeof(int) * len * len);
	err = hipMalloc(&final_d, sizeof(int) * final.size());
	err = hipMalloc(&finalIds_d, sizeof(int) * finalIds.size());
	err = hipMalloc(&freq_d, sizeof(int) * len * len);


	err = hipMemcpy(a_d, A_OW_112.data(), sizeof(int64_t) * A_OW_112.size(), hipMemcpyHostToDevice);
	err = hipMemcpy(b_d, B_OW_112.data(), sizeof(int64_t) * B_OW_112.size(), hipMemcpyHostToDevice);

	err = hipMemset(bedrock_d, 0, sizeof(int) * len * len);

	err = hipMemcpy(final_d, final.data(), sizeof(int) * final.size(), hipMemcpyHostToDevice);
	err = hipMemcpy(finalIds_d, finalIds.data(), sizeof(int) * finalIds.size(), hipMemcpyHostToDevice);


	std::tuple<size_t, int, int> best = { 0, 0, 0 };
	auto part_best = best;

	//1000 (*1000*1000) takes 7500ms before new shiny kernel
	//1000 (*1000*1000) takes 2500ms with new shinyish kernel
	//60 (*4096*4096) takes 2000ms with new shinyish kernel
	//15 (*8192*8192) takes 2100ms with new shinyish kernel
	//15 (*8192*8192) takes 2500ms with finished? kernel
	//15 (*8192*8192) takes 1900ms after neglecting to read freq_buf
	// ^ at this point after scaling up to 150 iters, we can do about 10^9 blocks per second
	//15 (*8192*8192) takes 6000ms after first unclean cuda trial
	//15 (*8192*8192) takes 1000ms after cleaned up cuda memory copys!
	// ^ at this point after scaling up to 150 iters, we can do about 2.5e9 blocks per second
	dim3 DimGrid(len / THREADSPERBLOCK_X, len / THREADSPERBLOCK_Y);
	dim3 DimBlock(THREADSPERBLOCK_X, THREADSPERBLOCK_Y);

	dim3 DimGrid2((len * len) / THREADSPERBLOCK_2);
	dim3 DimBlock2(THREADSPERBLOCK_2);

	for (int i = start; i < end; i++) {
		auto t1 = Clock::now();

		offset = { spiral(i)[0] * len , spiral(i)[1] * len };
		//std::cout << offset[0] << ' ' << offset[1] << std::endl;


		err = hipMemcpy(off_d, offset.data(), sizeof(int) * offset.size(), hipMemcpyHostToDevice);
		getBedrockTile << <DimGrid, DimBlock >> > (a_d, b_d, off_d, bedrock_d);

		//err = hipMemcpy(bedrock.data(), bedrock_d, sizeof(int) * bedrock.size(), hipMemcpyDeviceToHost);


		//begin labeling clumps
		err = hipMemset(labels_d, 0, sizeof(int) * len * len);

		Init << <DimGrid, DimBlock >> > (bedrock_d, labels_d);
		Analyze << <DimGrid, DimBlock >> > (labels_d);
		Reduce << <DimGrid, DimBlock >> > (bedrock_d, labels_d);
		Analyze << <DimGrid, DimBlock >> > (labels_d);
		//finish labeling clumps

		//err = hipMemcpy(bedrock.data(), bedrock_d, sizeof(int) * bedrock.size(), hipMemcpyDeviceToHost);
		//err = hipMemcpy(labels.data(), labels_d, sizeof(int) * labels.size(), hipMemcpyDeviceToHost);

		err = hipMemset(freq_d, 0, sizeof(int) * len * len);

		getFrequency << <DimGrid2, DimBlock2 >> > (labels_d, freq_d);

		reduction << <DimGrid2, DimBlock2 >> > (freq_d, final_d, finalIds_d);
		//err = hipMemcpy(freq.data(), freq_d, sizeof(int) * freq.size(), hipMemcpyDeviceToHost);
		err = hipMemcpy(final.data(), final_d, sizeof(int) * final.size(), hipMemcpyDeviceToHost);
		err = hipMemcpy(finalIds.data(), finalIds_d, sizeof(int) * finalIds.size(), hipMemcpyDeviceToHost);

		hipDeviceSynchronize();

		int record = 0, recordi = 0;
		for (int i = 0; i < final.size(); i++) {
			if (final[i] > record) {
				recordi = i;
				record = final[i];
			}
		}
		int recordX = finalIds[recordi] / len;
		int recordZ = finalIds[recordi] % len;

		auto t2 = Clock::now();

		if (!hasflag_q){
		std::cout << i << ' ';
		std::cout << ' ' << record << " @ (" << recordX + offset[0] << ", " << recordZ + offset[1] << ')' << "                             " << std::endl;
		}

		std::chrono::microseconds ms = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);
		int per_sec = (float)(1000000) / ms.count();

		if (hasflag_b){
			std::cout << per_sec << "tiles/s"  << " Tile#: " << i << '\r';
		} else {
			std::cout << per_sec << "tiles/s"  << " ETA: " << ms * (end - i) << " Tile#: " << i << '\r';
		}


		std::tuple<size_t, int, int> result = { record, recordX + offset[0], recordZ + offset[1] };

		if (std::get<0>(result) > std::get<0>(best)) {
			best = result;
		}
		if (hasflag_b){
			if (std::get<0>(result) > std::get<0>(part_best)) {
				part_best = result;
			}

			if (i % d == 0 && i != start){
				saveLog("recordFile.txt", part_start, part_end, part_best);
				part_start = i;
				part_end = i + d;
				part_best = {0, 0, 0};
			}
		}
	}

	std::cout << "Best found: " << "                             " << std::endl;
	std::cout << std::get<0>(best) << " @ (" << std::get<1>(best) << ", " << std::get<2>(best) << ')' << std::endl;

	hipFree(a_d);
	hipFree(b_d);
	hipFree(off_d);
	hipFree(bedrock_d);
	hipFree(freq_d);
	hipFree(final_d);
	hipFree(finalIds_d);
	hipFree(labels_d);

	if (!hasflag_b){
		saveLog("recordFile.txt", start, end, best);
	}
}
